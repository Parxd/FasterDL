#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "../utils/misc_utils.h"
#include "../utils/thrust_utils.h"
#include "../utils/cublas_utils.h"

using d_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    const d_type alpha = 1.0;
    const d_type beta = 0.0;
    
    const int l1 = 2;
    const int l2 = 2;
    thrust::host_vector<d_type> W1 = {0.15, 0.20, 0.25, 0.30}; // layer 1 weights
    thrust::host_vector<d_type> W2 = {0.40, 0.45, 0.50, 0.55}; // layer 2 weights
    thrust::host_vector<d_type> b1 = {0.35, 0.35}; 
    thrust::host_vector<d_type> b2 = {0.60, 0.60};

    thrust::host_vector<d_type> in = {0.05, 0.10};
    thrust::host_vector<d_type> out(2);

    // print_host_thrust(1, 2, in, 1);
    // print_host_thrust(l1, l2, W1, l1);
    thrust::device_vector<d_type> d_in = in;
    thrust::device_vector<d_type> d_W1 = W1;
    thrust::device_vector<d_type> d_W2 = W2;
    thrust::device_vector<d_type> d_b1 = b1;
    thrust::device_vector<d_type> d_b2 = b2;
    thrust::device_vector<d_type> d_imdte_out = out;
    thrust::device_vector<d_type> d_out(2);

    const int m = 2;
    const int n = 2;
    const int k = 1;

    CUBLAS_CHECK(
        hipblasDgemm(
            cublasH, 
            HIPBLAS_OP_N, 
            HIPBLAS_OP_N, 
            n, k, m, 
            &alpha, 
            thrust::raw_pointer_cast(&d_in[0]), n,
            thrust::raw_pointer_cast(&d_W1[0]), m, 
            &beta,
            thrust::raw_pointer_cast(&d_imdte_out[0]), n
        )
    );
    // hipDeviceSynchronize();  // don't need this since we're using cublas stream?
    CUBLAS_CHECK(
        hipblasDgeam(
            cublasH,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            m, n,
            &alpha,
            thrust::raw_pointer_cast(&d_imdte_out[0]), m,
            &beta,
            thrust::raw_pointer_cast(&b1[0]), n,
            thrust::raw_pointer_cast(&d_out[0]), m
        )
    );
    hipStreamSynchronize(stream);
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    print_device_thrust<d_type>(1, 2, d_out, 1);

    return EXIT_SUCCESS;
}