#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../include/cublas_utils.h"

using d_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    const d_type alpha = 1.0;
    const d_type beta = 0.0;

	

    return EXIT_SUCCESS;
}