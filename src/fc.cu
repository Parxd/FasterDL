#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../include/cublas_utils.h"

using d_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    const d_type alpha = 1.0;
    const d_type beta = 0.0;

    int m = 1;
    int n = 1;
    int k = 3;
    d_type W[m * k]{0.5, 0.5, 0.5};
    d_type X[k * n]{2,   2,   2  };
    d_type Z[m * n]{0};

    d_type* d_W;
    d_type* d_X;
    d_type* d_Z;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(W)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_X), sizeof(X)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Z), sizeof(Z)));
    CUDA_CHECK(hipMemcpyAsync(d_W, W, sizeof(W), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_X, X, sizeof(X), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(
        hipblasDgemm(
            cublasH, 
            HIPBLAS_OP_N, 
            HIPBLAS_OP_N, 
            m, n, k, 
            &alpha, 
            d_W, 1, 
            d_X, 3, 
            &beta, 
            d_Z, 1
        )
    );

    CUDA_CHECK(hipMemcpyAsync(Z, d_Z, sizeof(Z), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_Z));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    print_matrix(m, n, Z, 1);

    return EXIT_SUCCESS;
}