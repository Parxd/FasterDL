#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include "../include/cublas_utils.h"

using d_type = double;

void forward() {

}

void backward() {
	
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    const d_type alpha = 1.0;
    const d_type beta = 0.0;

    // Layer 1 - 3 nodes
    // Layer 2 - 4 nodes
    // Layer 3 - 2 nodes
    
    
    return EXIT_SUCCESS;
}