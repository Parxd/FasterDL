#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../utils/thrust_utils.h"

using d_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    const d_type alpha = 1.0;
    const d_type beta = 0.0;

    const int m = 3;
    const int n = 2;
    const int k = 2;
    d_type W[k * m]{1, 2, 3, 4, 5, 6};
    d_type X[m * n]{10, 11, 12, 13, 14, 15};
    d_type Z[k * n];

    d_type* d_W;
    d_type* d_X;
    d_type* d_Z;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_W), sizeof(W)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_X), sizeof(X)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_Z), sizeof(Z)));
    CUDA_CHECK(hipMemcpyAsync(d_W, W, sizeof(W), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_X, X, sizeof(X), hipMemcpyHostToDevice, stream));
    
    // Reverse GEMM to compute (row-major) transposed Z
    CUBLAS_CHECK(
        hipblasDgemm(
            cublasH, 
            HIPBLAS_OP_N, 
            HIPBLAS_OP_N, 
            n, k, m, 
            &alpha, 
            d_X, n,
            d_W, m, 
            &beta,
            d_Z, n
        )
    );

    CUDA_CHECK(hipMemcpyAsync(Z, d_Z, sizeof(Z), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_Z));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    print_matrix(k, n, Z, k);

    return EXIT_SUCCESS;
}